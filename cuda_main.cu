#include "hip/hip_runtime.h"
#include "common_functions.h"  // Include common functions
#include "Cluster.h"            // Include the Cluster class
#include <iostream>
#include <fstream>
#include <chrono>               // For time measurement

// Device function to calculate squared Euclidean distance on the GPU
__device__ double squared_euclidean_distance_device(const Point& pt, const Cluster& cl) {
    return pow(pt.get_x() - cl.get_x(), 2) + pow(pt.get_y() - cl.get_y(), 2);
}

// Kernel function to assign points to the nearest cluster (on the GPU)
__global__ void assign_points(Point* points, Cluster* clusters, int num_points, int num_clusters) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < num_points) {
        double min_dist = squared_euclidean_distance_device(points[i], clusters[0]);
        int nearest_cluster_id = 0;

        for (int j = 1; j < num_clusters; ++j) {
            double dist = squared_euclidean_distance_device(points[i], clusters[j]);
            if (dist < min_dist) {
                min_dist = dist;
                nearest_cluster_id = j;
            }
        }
        points[i].set_id(nearest_cluster_id);
    }
}

// K-means algorithm with CUDA, running 20 iterations
void kmeans_cuda(std::vector<Point>& points, std::vector<Cluster>& clusters, double& total_time) {
    const int max_iterations = 20;
    int iterations = 0;

    // Allocate memory on the GPU
    Point* d_points;
    Cluster* d_clusters;
    hipMalloc(&d_points, points.size() * sizeof(Point));
    hipMalloc(&d_clusters, clusters.size() * sizeof(Cluster));

    // Copy data to the GPU
    hipMemcpy(d_points, points.data(), points.size() * sizeof(Point), hipMemcpyHostToDevice);
    hipMemcpy(d_clusters, clusters.data(), clusters.size() * sizeof(Cluster), hipMemcpyHostToDevice);

    // Set up CUDA block and grid sizes
    int blockSize = 256;
    int numBlocks = (points.size() + blockSize - 1) / blockSize;

    while (iterations < max_iterations) {
        auto start_iter = std::chrono::high_resolution_clock::now();

        // Step 1: Assign points to the nearest cluster (on GPU)
        assign_points<<<numBlocks, blockSize>>>(d_points, d_clusters, points.size(), clusters.size());

        // Copy updated points back to host (CPU)
        hipMemcpy(points.data(), d_points, points.size() * sizeof(Point), hipMemcpyDeviceToHost);

        // Step 2: Update cluster centroids (sequential step on CPU)
        for (auto& cluster : clusters) {
            cluster.delete_values(); // Reset values
        }

        for (const auto& point : points) {
            clusters[point.get_cluster_id()].add_point(point);
        }

        for (auto& cluster : clusters) {
            cluster.update_values();
        }

        // Copy updated clusters back to GPU
        hipMemcpy(d_clusters, clusters.data(), clusters.size() * sizeof(Cluster), hipMemcpyHostToDevice);

        auto end_iter = std::chrono::high_resolution_clock::now();
        double iteration_time = std::chrono::duration<double>(end_iter - start_iter).count();  // Time in seconds
        total_time += iteration_time;  // Add to total time

        iterations++;
    }

    // Free GPU memory
    hipFree(d_points);
    hipFree(d_clusters);

    std::cout << "Completed " << max_iterations << " iterations on the GPU.\n";
}

// Function to run the K-means algorithm with CUDA and save results to a file
void run_kmeans_cuda(int num_points, int num_clusters, std::ofstream& output_file) {
    std::cout << "Running K-means with " << num_points << " points and " << num_clusters << " clusters on the GPU.\n";

    // Generate points and initialize clusters
    std::vector<Point> points = generate_points(num_points);
    std::vector<Cluster> clusters = initialize_clusters(num_clusters, points);

    // Measure total execution time
    double total_time = 0.0;

    kmeans_cuda(points, clusters, total_time);

    // Calculate time per iteration
    double time_per_iteration = total_time / 20.0;  // We run exactly 20 iterations

    // Save the result to the output file
    output_file << "Configuration: " << num_points << " points, " << num_clusters << " clusters (CUDA)\n";
    output_file << "Total execution time: " << total_time << " seconds\n";
    output_file << "Time per iteration: " << time_per_iteration << " seconds\n";
    output_file << "----------------------------------------\n";
}

int main() {
    // Open file to save results
    std::ofstream output_file("cuda_main_results.txt");
    if (!output_file.is_open()) {
        std::cerr << "Error: Could not open the file to save results.\n";
        return 1;
    }

    // Define the different configurations for normal runs
    std::vector<int> num_points_list = {100000, 250000, 500000, 1000000};
    std::vector<int> num_clusters_list = {5, 10, 20};

    // Run the algorithm for each configuration and save results
    for (int num_points : num_points_list) {
        for (int num_clusters : num_clusters_list) {
            run_kmeans_cuda(num_points, num_clusters, output_file);
        }
    }

    // Special run for 1 million points with larger cluster numbers
    std::vector<int> large_cluster_list = {100, 250, 500, 1000};
    output_file << "Running special CUDA K-means test for 1 million points with larger cluster counts.\n";
    for (int num_clusters : large_cluster_list) {
        run_kmeans_cuda(1000000, num_clusters, output_file);
    }

    // Close the file
    output_file.close();

    return 0;
}
